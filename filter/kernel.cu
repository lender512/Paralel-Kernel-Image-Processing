#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <stdint.h>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>



#define from2Dto1D(i, j) (((i) * gridDim.x) + (j))
__global__ void betterCombKernel(const unsigned char* source, unsigned char* target, const int* kernel, int kernelDim, int divisor)
{

    //int kernelDim = 5;
    int x = blockIdx.x + kernelDim / 2;
    int y = blockIdx.y + kernelDim / 2;

    
    int idx = x + y * gridDim.x;
    int value = 0;

    for (int i = 0; i < kernelDim; ++i)
    {
        for (int j = 0; j < kernelDim; ++j) {
            value += kernel[i * kernelDim + j] * source[from2Dto1D(x + (i - kernelDim / 2), y + (j - kernelDim / 2))];
        }
    }

    value = value / (divisor);
    if (value < 255) {
        if (value < 0) {
            target[idx] = 0;
        }
        else {
            target[idx] = value;
        }
    }
    else {
        target[idx] = 255;
    }
    
}


int main() {

    std::string folder = "C:/Users/luis/Desktop/source/";

    cv::Mat img = cv::imread(folder + "input.jpg");



    cv::Mat filterImgMat(img);
    unsigned char* sourceR;
    unsigned char* sourceG;
    unsigned char* sourceB;
    unsigned char* targetR;
    unsigned char* targetG;
    unsigned char* targetB;
    int* kernel;
    
    const int DIM = img.size().height;
    dim3 grid(DIM, DIM);


    /*int kernelH[] = { 1, 4, 6, 4, 1,
                     4, 16, 24, 16, 4,
                     2, 24, 36, 24, 2,
                     4, 16, 24, 16, 4,
                     1, 4, 6, 4, 1};*/

    /*int kernelH[] = {
        0, 0, 1, 2, 1, 0, 0,
        0, 3, 13, 22, 13, 3, 0,
        1, 13, 59, 97, 59, 13, 1,
        2, 22, 97, 159, 97, 22, 2,
        1, 13, 59, 97, 59, 13, 1,
        0, 3, 13, 22, 13, 3, 0,
        0, 0, 1, 2, 1, 0, 0,

    };*/


    int kernelH[] = { -1, 0, 1,
                      -1, 0, 1,
                      -1, 0, 1 };

    int kernelDim = sqrt(sizeof(kernelH)/sizeof(int));
    int divisor = 1;

    hipMalloc((void**)&kernel, kernelDim * kernelDim * sizeof(int));


    hipMalloc((void**)&sourceR, DIM * DIM * sizeof(char));
    hipMalloc((void**)&sourceG, DIM * DIM * sizeof(char));
    hipMalloc((void**)&sourceB, DIM * DIM * sizeof(char));
    hipMalloc((void**)&targetR, (DIM) * (DIM) * sizeof(char));
    hipMalloc((void**)&targetG, (DIM) * (DIM) * sizeof(char));
    hipMalloc((void**)&targetB, (DIM) * (DIM) * sizeof(char));



    uchar* imgMatrixR = new uchar[DIM * DIM];
    uchar* imgMatrixG = new uchar[DIM * DIM];
    uchar* imgMatrixB = new uchar[DIM * DIM];

    for (int col = 0; col < DIM; ++col) {
        for (int row = 0; row < DIM; ++row) {
            auto v = img.at<cv::Vec3b>(col, row);
            imgMatrixR[DIM * col + row] = v[0];
            imgMatrixG[DIM * col + row] = v[1];
            imgMatrixB[DIM * col + row] = v[2];
        }
    }

    hipMemcpy(kernel, kernelH, kernelDim * kernelDim * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(sourceR, imgMatrixR, DIM * DIM * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(sourceG, imgMatrixG, DIM * DIM * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(sourceB, imgMatrixB, DIM * DIM * sizeof(char), hipMemcpyHostToDevice);
    
 

    betterCombKernel << < grid, 1 >> > (sourceR, targetR, kernel, kernelDim, divisor);
    betterCombKernel << < grid, 1 >> > (sourceG, targetG, kernel, kernelDim, divisor);
    betterCombKernel << < grid, 1 >> > (sourceB, targetB, kernel, kernelDim, divisor);


    hipMemcpy(imgMatrixR, targetR, DIM * DIM * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(imgMatrixG, targetG, DIM * DIM * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(imgMatrixB, targetB, DIM * DIM * sizeof(char), hipMemcpyDeviceToHost);

    for (int col = kernelDim/2; col < DIM; ++col) {
        for (int row = kernelDim/2; row < DIM; ++row) {
            cv::Vec3b v = filterImgMat.at<cv::Vec3b>(col, row);
            v[0] = imgMatrixR[DIM * row + col];
            v[1] = imgMatrixG[DIM * row + col];
            v[2] = imgMatrixB[DIM * row + col];
            filterImgMat.at<cv::Vec3b>(col, row) = v;
        }
    }

    cv::imwrite(folder + "output5.jpg", filterImgMat);
    
    hipFree(sourceR);
    hipFree(sourceG);
    hipFree(sourceB);
    hipFree(targetR);
    hipFree(targetG);
    hipFree(targetB);
    hipFree(kernel);
    delete [] imgMatrixR;
    delete [] imgMatrixG;
    delete [] imgMatrixB;

}